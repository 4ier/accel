
#include <hip/hip_runtime.h>
__device__ int f(int a) {
  return 2 * a;
}

__global__ void add(const int a[], const int b[], int c[]) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    c[i] = f(a[i] + b[i]);
}
